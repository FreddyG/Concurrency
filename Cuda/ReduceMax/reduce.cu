#include "hip/hip_runtime.h"
/*
 * simulate.c
 *
 * Implement your (parallel) simulation here!
 */

#include <stdio.h>
#include <stdlib.h>
#include <pthread.h>
#include <string.h>

#include "reduce.h"

#include <iostream>

#define THREADS_PER_BLOCK 512

using namespace std;

/* Utility function, use to do error checking.

   Use this function like this:

   checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));

   And to check the result of a kernel invocation:

   checkCudaCall(hipGetLastError());
*/
static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        cerr << "cuda error: " << hipGetErrorString(result) << endl;
        exit(1);
    }
}

// Returns the next power of 2 that is >= n
__device__ int nextPowOfTwo(int n) {
    // 0 is also a power of 2
    if (n == 0) {
        return n;
    }

    else {
        int pow = 1;
        while ( pow < n ) {
            n *= 2;
        }

        return pow;
    }
}

// Finds the minimum value in an array
__global__ void reduceKernel(double *array, int N, double *out)
{
    // Reduction (min/max/avr/sum), works for any blockDim.x:
    int thread2;
    double temp;
    __shared__ double min[THREADS_PER_BLOCK];

	// Total number of threads, rounded up to the next power of two
    int nTotalThreads = nextPowOfTwo(blockDim.x);

    // tree-wise reduction
    while(nTotalThreads > 1)
    {
        int halfPoint = (nTotalThreads / 2);	// divide by two
        // only the first half of the threads will be active.

        if (threadIdx.x < halfPoint)
        {
            thread2 = threadIdx.x + halfPoint;

            // make sure we're not dealing with a non-existent thread
            if (thread2 < blockDim.x)
            {
                // Get the shared value stored by another thread
                temp = min[thread2];
                if (temp < min[threadIdx.x])
                    min[threadIdx.x] = temp; 
            }
        }
        __syncthreads();

        // Reducing the binary tree size by two:
        nTotalThreads = halfPoint;
    }
}

// return the mimum value of a given array
double reduce_min(double *array, int N)
{
    // allocate the array and result pointer on the GPU
    double* dev_array = NULL;
    double* dev_result = NULL;

    checkCudaCall(hipMalloc((void **) &dev_array, N * sizeof(double)));
    if (dev_array == NULL) {
        cout << "could not allocate memory!" << endl;
        exit(1);
    }

    checkCudaCall(hipMalloc((void **) &dev_result, sizeof(double)));
    if (dev_result == NULL) {
        cout << "could not allocate memory!" << endl;
        hipFree(&dev_array);
        exit(1);
    }

    // copy the data to the GPU
    checkCudaCall(hipMemcpy(dev_array, array, N*sizeof(double), hipMemcpyHostToDevice));

    // calculate the amount of blocks to be used
    // The first kernel invocation should reduce the array to one that can be
    // reduced by one block, so there should be THREADS_PER_BLOCK elements left
    int blocks = (N / 2) / THREADS_PER_BLOCK;

    reduceKernel <<< blocks, THREADS_PER_BLOCK >>>
        (dev_array, N, dev_result);

    // copy the result back to the main program
    double result[1];
    checkCudaCall(hipMemcpy(result, dev_result, sizeof(double), hipMemcpyDeviceToHost));

    return result[0];
}
