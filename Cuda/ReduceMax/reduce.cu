#include "hip/hip_runtime.h"
/*
 * simulate.c
 *
 * Implement your (parallel) simulation here!
 */

#include <stdio.h>
#include <stdlib.h>
#include <pthread.h>
#include <string.h>

#include "reduce.h"

#include <iostream>

#define THREADS_PER_BLOCK 512

using namespace std;

/* Utility function, use to do error checking.

   Use this function like this:

   checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));

   And to check the result of a kernel invocation:

   checkCudaCall(hipGetLastError());
*/
static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        cerr << "cuda error: " << hipGetErrorString(result) << endl;
        exit(1);
    }
}

// Returns the next power of 2 that is >= n
__device__ int nextPowOfTwo(int n) {
    // 0 is also a power of 2
    if (n == 0) {
        return n;
    }

    else {
        int pow = 1;
        while ( pow < n ) {
            n *= 2;
        }

        return pow;
    }
}

// Finds the minimum value in an array
__global__ void reduceKernel(double *array, int N, double *out)
{
    // each thread handles a chunk of the array, and writes it to block-shared
    // memory, reducing the array to a new array with THREADS_PER_BLOCK
    // elements.
    __shared__ double min_per_thread[THREADS_PER_BLOCK];
    
    int stepsize = N / THREADS_PER_BLOCK;

    int start = threadIdx.x * stepsize,
        end   = start + stepsize;

    double min = array[start];
    for (int i = start + 1; i < end; ++i) {
        if (array[i] < min) {
            min = array[i];
        }
    }

    min_per_thread[threadIdx.x] = min;
    __syncthreads();

    // one of the threads performs a further reduction step
    min = min_per_thread[0];
    if (threadIdx.x == 0) {
        for (int i = 1; i < THREADS_PER_BLOCK; ++i) {
            if (min_per_thread[i] < min) {
                min = min_per_thread[i];
            }
        }
    }

    out[0] = min;
}

// return the mimum value of a given array
double reduce_min(double *array, int N)
{
    // allocate the array and result pointer on the GPU
    double* dev_array = NULL;
    double* dev_result = NULL;

    checkCudaCall(hipMalloc((void **) &dev_array, N * sizeof(double)));
    if (dev_array == NULL) {
        cout << "could not allocate memory!" << endl;
        exit(1);
    }

    checkCudaCall(hipMalloc((void **) &dev_result, sizeof(double)));
    if (dev_result == NULL) {
        cout << "could not allocate memory!" << endl;
        hipFree(&dev_array);
        exit(1);
    }

    // copy the data to the GPU
    checkCudaCall(hipMemcpy(dev_array, array, N*sizeof(double), hipMemcpyHostToDevice));

    reduceKernel <<< 1, THREADS_PER_BLOCK >>>
        (dev_array, N, dev_result);

    // copy the result back to the main program
    double result[1];
    checkCudaCall(hipMemcpy(result, dev_result, sizeof(double), hipMemcpyDeviceToHost));

    return result[0];
}
